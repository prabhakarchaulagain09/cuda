
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void helloCUDA() {
    printf("Hello from the GPU!\n");
}

int main() {
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}

